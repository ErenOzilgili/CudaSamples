#include <hip/hip_runtime.h>
#include <iostream>

//Width and Height are 16 respectively
#define BLOCK_SIZE 16

struct Matrix{
	int* elementPtr;
	int width;
	int stride; 
	//Pitch is stride in bytes, essentially.
	//Stride is how many elements has been allocated on that row with padding
	//while pitch is strides in bytes per row mentioned as above.
	//In this example, we will be using width = stride.
	int height;
};

__device__ void setElement(Matrix mat, int column, int row, int value){
	if(column < mat.width && row < mat.height){
		*(mat.elementPtr + row * mat.stride + column) = value;
	}
}

__device__ int getElement(Matrix mat, int column, int row){
	return *(mat.elementPtr + row * mat.stride + column);
}

__device__ Matrix returnSubMat(Matrix main, int subBlockColumn, int subBlockRow){
	Matrix sub;
	sub.width = BLOCK_SIZE;
	sub.height = BLOCK_SIZE;
	sub.stride = main.stride;
	sub.elementPtr = &main.elementPtr[ main.stride * subBlockRow * BLOCK_SIZE
									     + subBlockColumn * BLOCK_SIZE];
	/*  subBlockRow * BLOCK_SIZE determines which row we should reach.
		Multiplying above by main.stride travels elements in row-major ordering
		to reach to the first element in our subBlock's row.
		After adding subBlockColumn * BLOCK_SIZE, we are now in the first thread of our block.
		We place the pointer here.
	*/
	/*
	 	4 x 4 matrix

		|--|--|  --> Here |--| symbolizes a 2 X 2 thread block.
		|--|--|			  |--|
		|--|/*|
		|--|**|  --> Total of 4 thread blocks

		In order to make our pointer point to /
		we do the above procedure.	
	*/

	return sub;
}

//Method declerations for usage from main method
int* matrixMul(const Matrix A, const Matrix B, Matrix C);
__global__ void matMul(const Matrix, const Matrix, Matrix);

//Generate sample matrices. Use it in CPU -- Host
void generateMatrix(Matrix& mat, int value){
	for(int iCol = 0; iCol < mat.width; iCol++){
		for(int iRow = 0; iRow < mat.height; iRow++){
			*(mat.elementPtr + iRow * mat.stride + iCol) = value;
		}
	}
}

int main(){
	//Create the sample arrays
	Matrix A;
	A.width = 64;
	A.stride = 64;
	A.height = 64;

	//Allocate space for samples
	size_t size = A.width * A.height * sizeof(int);
	A.elementPtr = (int*)malloc(size);

	//Generate with random values
	int valueA = 2;
	generateMatrix(A, valueA);

	Matrix B;
	B.width = 64;
	B.stride = 64;
	B.height = 64;

	//Allocate space for samples
	size = B.width * B.height * sizeof(int);
	B.elementPtr = (int*)malloc(size);

	//Generate with random values
	int valueB = 3;
	generateMatrix(B, valueB);

	//Empty matrix result C
	Matrix C;
	C.width = 64;
	C.stride = 64;
	C.height = 64;

	//Allocate space for samples
	size = C.width * C.height * sizeof(int);
	C.elementPtr = (int*)malloc(size);

	std::cout << "Generated A , B and C in main()" << std::endl;

	std::cout << "Entering matrixMul()" << std::endl;
	int* resultPtr_C = matrixMul(A, B ,C);

	std::cout << "Exited matrixMul, kernel should have finished. Stopping the program after printing a value" << std::endl;
	std::cout << "First index of the resulting C matrix is: " << *resultPtr_C << std::endl;
	std::cout << "All indexes of the A is: " << *A.elementPtr << std::endl;
	std::cout << "All indexes of the B is: " << *B.elementPtr << std::endl;
	std::cout << "Expected value for all the indexes of resulting C was: " << valueA * valueB * 64 << std::endl;
	return 0;
}

int* matrixMul(const Matrix A, const Matrix B, Matrix C){
	//Load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.stride = A.stride;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(int);

	//Allocate space in device memory
	hipMalloc(&d_A.elementPtr, size);
	//Transfer (copy) the data to device
	hipMemcpy(d_A.elementPtr, A.elementPtr, size, hipMemcpyHostToDevice);

	//Load B to device memory
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	d_B.stride = B.stride;
	size = B.height * B.width * sizeof(int);

	hipMalloc(&d_B.elementPtr, size);
	hipMemcpy(d_B.elementPtr, B.elementPtr, size, hipMemcpyHostToDevice);

	//Prepare a space for the resulting matrx C, allocate memory
	Matrix d_C;
	d_C.width = C.width;
	d_C.stride = C.stride;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(int);

	hipMalloc(&d_C.elementPtr, size);

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

	matMul<<< dimGrid, dimBlock >>>(d_A, d_B, d_C);

	//Copy back the resulting C matrix
	hipMemcpy(C.elementPtr, d_C.elementPtr, size, hipMemcpyDeviceToHost);

	//CLear the device memory pointers allocation
	hipFree(d_A.elementPtr);
	hipFree(d_B.elementPtr);
	hipFree(d_C.elementPtr);

	//Return the pointer pointing to the first index of the matrix --> Just for checking
	return C.elementPtr;
}

__global__ void matMul(const Matrix A, const Matrix B, Matrix C){
	int subBlockColumn = blockIdx.x;
	int subBlockRow = blockIdx.y;

	Matrix subC = returnSubMat(C, subBlockColumn, subBlockRow);

	int accMul_C = 0;

	int tidX = threadIdx.x;
	int tidY = threadIdx.y;

	for(int m = 0; m < A.width / BLOCK_SIZE ; m++){
		//Return the sub matrices for tiled multiplication
		Matrix subA = returnSubMat(A, m, subBlockRow);
		Matrix subB = returnSubMat(B, subBlockColumn, m);

		//Create a shared memory (scratcpad memory) for shared loading
		//This is a shared memory for a thread block
		//Threads within thread block will work together to get the values from DRAM
		//However, each thread within threadblock will calculate the portion of their own sum after synching.
		__shared__ int A_vals[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ int B_vals[BLOCK_SIZE][BLOCK_SIZE];

		//Load to the shared memory individual elements.
		A_vals[tidY][tidX] = getElement(subA, tidX, tidY);
		B_vals[tidY][tidX] = getElement(subB, tidX, tidY);

		__syncthreads(); //Synchronise before calculating the accMul_C

		for(int i = 0; i < BLOCK_SIZE; i++){
			accMul_C += A_vals[tidY][i] * B_vals[i][tidX];
		}

		__syncthreads(); //Synchronise before preceeding into other iteration
	}

	//Record the resulting multplication into the one thread in subC
	setElement(subC, tidX, tidY, accMul_C);
}